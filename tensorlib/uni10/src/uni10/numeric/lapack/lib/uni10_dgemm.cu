#include <uni10/numeric/lapack/uni10_lapack.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"
//#include "hipsolver.h"

namespace uni10{
void getRows(int M, int N, int start, int span, double* iA, double* fA, mmtype how);
void getCols(int M, int N, int start, int span, double* iA, double* fA, mmtype how);
void putBack(int startR, int startC, int spanR, int spanC, int M, int N, double *subA, double *A, mmtype how);
void uni10Dgemm(int p, int q, int M, int N, int K, double* A, double* B, double* C, mmtype how){
	int pM = M / p;
	int qN = N / q;
	int rows, cols;
	rows = pM;// + (M % p);
	cols = qN;// + (N % q);
	double *subA;
	double *subB;
	double *subC;
	double alpha = 1.0;
	double beta = 0.0;
	hipblasStatus_t status;
	hipblasHandle_t handle;
	status = hipblasCreate(&handle);
	if(how & 4)
		assert(hipMalloc((void**)&subA, rows * K * sizeof(double)) == hipSuccess);
	if(how & 2 || q > 1)
		assert(hipMalloc((void**)&subB, K * cols * sizeof(double)) == hipSuccess);
	if(how & 1 || q > 1)
		assert(hipMalloc((void**)&subC, rows * cols * sizeof(double)) == hipSuccess);
	int rChunkNum = (M + pM - 1) / pM;
	int cChunkNum = (N + qN - 1) / qN;
	for(int i = 0; i < rChunkNum; i++){
		for(int j = 0; j < cChunkNum; j++){
			rows = pM; // + (M % p) * ((i + 1) / p);
			cols = qN; // + (N % q) * ((j + 1) / q);
			if(i == rChunkNum - 1 && (M % pM) > 0)
				rows = M % pM ;
			if(j == cChunkNum - 1 && (N % qN) > 0)
				cols = N % qN;
			if(how & 4)
				getRows(M, K, i * pM, rows, A, subA, how);
			else
				subA = A + i * pM * K;
			if(how & 2 || q > 1)
				getCols(K, N, j * qN, cols, B, subB, how);
			else
				subB = B;
			if(how & 1 || q > 1){
				status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, cols, rows, K, &alpha, subB, cols, subA, K, &beta, subC, cols);
				putBack(i * pM, j * qN, rows, cols, M, N, subC, C, how);
			}
			else{
				subC = C + i * pM * N;
				status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, cols, rows, K, &alpha, subB, cols, subA, K, &beta, subC, cols);
			}
			//hipblasStatus_t status = cublasGetError();
			assert(status == HIPBLAS_STATUS_SUCCESS);
		}
	}
	if(how & 4)
		hipFree(subA);
	if(how & 2 || q > 1)
		hipFree(subB);
	if(how & 1 || q > 1)
		hipFree(subC);
}

void putBack(int startR, int startC, int spanR, int spanC, int M, int N, double *subC, double *C, mmtype how){
	if(how & 1){
		double* tmp = (double*)malloc(spanR * spanC * sizeof(double));
		assert(hipMemcpy(tmp, subC, sizeof(double) * spanR * spanC, hipMemcpyDeviceToHost) == hipSuccess);
		for(int pos = 0; pos < spanR; pos++)
			memcpy(C + (pos + startR) * N + startC, tmp + pos * spanC, spanC * sizeof(double));
		free(tmp);
	}
	else{
		for(int pos = 0; pos < spanR; pos++)
			assert(hipMemcpy(C + (pos + startR) * N + startC, subC + pos * spanC, spanC * sizeof(double), hipMemcpyDeviceToDevice) == hipSuccess);
	}
}

void getRows(int M, int N, int start, int span, double* iA, double* fA, mmtype how){
	assert(start + span <= M);
	if(how & 4){
		assert(hipMemcpy(fA, iA + start * N, sizeof(double) * N * span, hipMemcpyHostToDevice) == hipSuccess);
	}
}

void getCols(int M, int N, int start, int span, double* iB, double* fB, mmtype how){
	assert(start + span <= N);
	if(how & 2){
		double* tmp = (double*)malloc(M * span * sizeof(double));
		for(int i = 0; i < M; i++)
			memcpy(tmp + i * span, iB + i * N + start, span * sizeof(double));
		assert(hipMemcpy(fB, tmp, sizeof(double) * M * span, hipMemcpyHostToDevice) == hipSuccess);
		free(tmp);
	}
	else{
		for(int i = 0; i < M; i++)
			hipMemcpy(fB + i * span, iB + i * N + start, span * sizeof(double), hipMemcpyDeviceToDevice);
	}
}
};	/* namespace uni10 */
